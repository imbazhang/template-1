#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>

#include"hip/hip_runtime.h"
#include"hip/hip_runtime_api.h"
using namespace std;

int n, m, p, q;
const int szB = 3550, szL = 150;

double *matrix, *kernel, *result;

inline int read() {
    char c = getchar();
    int x = 0, f = 1;
    while (isdigit(c))
        x = (x << 3) + (x << 1) + (c^48), c = getchar();
    return x * f;
}

inline void read(double &r)
{
    double x=0,t=0;int s=0,f=1;char c=getchar();//x代表整数部分,t代表小数部分
    for (;!isdigit(c);c=getchar())
    {
        if (c=='-') f=-1;//读到负号就改变之
        if (c=='.') goto readt;//看到小数点,直接读小数部分
        if (c == ',') return ;
    }
    for (;isdigit(c)&&c!='.';c=getchar()) x=x*10+c-'0';//整数部分
    readt:for (;c=='.';c=getchar());//跳过小数点
    for (;isdigit(c);c=getchar()) t=t*10+c-'0',++s;//读小数部分,s代表小数有几位
    r=(x+t/pow(10,s))*f;//t除以10的s次方后变成小数部分
}

void input() {
    freopen("input.txt", "r", stdin);
    n = read();m=read();read();p = read();q = read();read();
//    cout << n << ' ' << m << ' ' << p << ' ' << q << endl;
    for (int i = 0; i < n; i++){
        for (int j = 0; j < m; j++) {
            read(matrix[i * szB + j]);
//            printf("%.3f ", matrix[i*szB+j]);
        }
//        printf("\n");
//            scanf("%lf,", &matrix[i*szB+j]);
//        scanf("%lf\n", &matrix[i*szB+m-1]);
    }
    for (int i = 0; i < p; i++){
        for (int j = 0; j < q; j++)
            read(kernel[i*szL+j]);
//            scanf("%lf,", &kernel[i*szL+j]);
//        scanf("%lf\n", &kernel[i*szL+q-1]);
    }
//    printf("input done\n");
}


void output() {
    FILE* fp = fopen("output.txt", "w");
    for (int i = 0; i < n; i++){
        for (int j = 0; j < m-1; j++)
            fprintf(fp, "%.3f,", result[(i+(p-1)/2)*szB+j+(q-1)/2]);
        fprintf(fp, "%.3f\n", result[(i+(p-1)/2)*szB+m-1+(q-1)/2]);
    }
    fclose(fp);
}

int getThreadNum()
{
    hipDeviceProp_t prop;
    int count;
    hipGetDeviceCount(&count);
    hipGetDeviceProperties(&prop, 0);
    return prop.maxThreadsPerBlock;
}

__global__ void conv(double *matrix, double *kernel, double *result, int n, int m, int p, int q){
    int ti = threadIdx.x;
    int bi = blockIdx.x;
    int id = (bi * blockDim.x + ti);
    if(id < (n+p-1) * (m+q-1)){
        int i = id / (m+q-1);
        int j = id % (m+q-1);
        double tmp = 0.0;
        for(int k = max(0, i-p+1); k <= i; k++)
            for(int l = max(0, j-q+1); l <= j; l++)
                tmp += matrix[k*szB+l] * kernel[(i-k)*szL + j-l];
        result[i*szB+j] = tmp;
    }
}


int main()
{
    double *matrixGpu;
    double *kernelGpu;
    double *resultGpu;

    matrix = (double*)malloc(sizeof(double)*szB*szB);
    result = (double*)malloc(sizeof(double)*szB*szB);
    kernel = (double*)malloc(sizeof(double)*szL*szL);

    hipMalloc((void**)&matrixGpu, szB*szB*sizeof(double));
    hipMalloc((void**)&kernelGpu, szL*szL*sizeof(double));
    hipMalloc((void**)&resultGpu, szB*szB*sizeof(double));

    input();
    hipMemcpy(matrixGpu, matrix, 3550 *3550 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(kernelGpu, kernel, 150*150*sizeof(double), hipMemcpyHostToDevice);

    int threadNum = getThreadNum();
    int blockNum = ((n+p-1) * (m+q-1) - 0.5) / threadNum + 1;
    
    conv<<<blockNum, threadNum>>>(matrixGpu, kernelGpu, resultGpu, n, m, p, q);
    hipMemcpy(result, resultGpu, 3550*3550 * sizeof(double), hipMemcpyDeviceToHost);

    output();
    hipFree(matrixGpu);
    hipFree(kernelGpu);
    hipFree(resultGpu);
    
    return 0;
}
